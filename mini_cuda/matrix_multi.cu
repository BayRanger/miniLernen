/**
 * basic code to cun cuda multiplication
 * 
 * TODO: benchmark the time duration 
 * 
 * this code is enough  to learn key cuda operation and data structure.
 * 
 * https://users.wfu.edu/choss/CUDA/docs/Lecture%205.pdf
*/


#define N 16

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void matrixMultCuda (int *a, int *b, int *c, int width) 
{
     int k, sum = 0;
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    if(col < width && row < width) 
    {
        for (k = 0; k < width; k++)
        sum += a[row * width + k] * b[k * width + col];
    
    c[row * width + col] = sum;
        
    }
}


void matrixMult (int a[N][N], int b[N][N], int c[N][N], int width)
{
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) 
        {
            int sum = 0;
            for (int k = 0; k < width; k++)
            {
                int m = a[i][k];
                int n = b[k][j];
                sum += m * n;
            }
            c[i][j] = sum;
        }
    }
}
int main() {

    int a[N][N], b[N][N], c[N][N];
    int *dev_a, *dev_b, *dev_c;
    // initialize matrices a and b with appropriate values
    int size = N * N * sizeof(int);
    //allocate memory in GPU
    hipMalloc((void **) &dev_a, size);
    hipMalloc((void **) &dev_b, size);
    hipMalloc((void **) &dev_c, size);
    //copy data from cpu to gpu
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    //definme the grid dimension : how many blocks a grid has
    //define the block dimension : how many threads f block has
    dim3 dimGrid(1, 1);
    dim3 dimBlock(N, N);

    //this kernel!
    matrixMultCuda<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);
    //perform calcualtion and copy back data to host
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    //free gpu memory acclocation
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
    //cpu version matrix multi
    matrixMult(a, b, c, N);
}
